#include "hip/hip_runtime.h"

#include <stdio.h>

#include "hip/hip_runtime.h"
#include ""
#include "timer.h"
#include "Timer1.h"


#define PRINTC_CUDA_ATOMIC_SHARED

#define XSIZE 1024
#define YSIZE 1024
#define HISTOBINS 8
hipError_t histoWithCuda(unsigned int* c, unsigned int* a, int xSize, int ySize);
hipError_t histoWithCudaAtomic(unsigned int* c, unsigned int* a, int xSize, int ySize);
hipError_t histoSharedMem(unsigned int* hIn, unsigned int* hOut, int xSize, int ySize);

void histoWithC(unsigned int* c, unsigned int* histo, int xSize, int ySize)
{
	int temp;
	for (int i = 0; i < ySize; i++)
		for (int j = 0; j < xSize; j++) {
			temp = c[i * xSize + j] % HISTOBINS;
			histo[temp]++;
		}
}


__global__ void histoKernel(unsigned int* c, unsigned int* a, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;
	int index = j * gridDim.x * blockDim.x + i;
	int temp;
	if (index < size) {
		temp = c[index] % HISTOBINS;
		a[temp] = a[temp] + 1;
	}

}

__global__ void histoAtomicKernel(unsigned int* c, unsigned int* a, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;
	int index = j * gridDim.x * blockDim.x + i;
	int temp;
	if (index < size) {
		temp = c[index] % HISTOBINS;
		atomicAdd(&a[temp], 1);
	}

}


__global__ void histoAccm(unsigned int* c, unsigned int* a, int size)
{
	int j = threadIdx.x;
	for (int i = j; i < size; i+=HISTOBINS) {
		a[j] += c[i];
	}
}

//************************************ HW11 **************************************************
__global__ void histoSharedKernel(unsigned int* c, unsigned int* a, int xSize, int ySize)
{
	// Shared memory allocation
	__shared__ unsigned int sharedHist[HISTOBINS];

	// Initialize shared memory histogram bins to 0
	int tid = threadIdx.x + threadIdx.y * blockDim.x;
	if (tid < HISTOBINS) {
		sharedHist[tid] = 0;
	}
	__syncthreads();

	// Calculate global index
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;

	if (i < xSize && j < ySize) {
		int index = j * xSize + i;
		int temp = c[index] % HISTOBINS;
		atomicAdd(&sharedHist[temp], 1);
	}
	__syncthreads();

	// Add shared histogram bins to global memory histogram
	if (tid < HISTOBINS) {
		atomicAdd(&a[tid], sharedHist[tid]);
	}
}


int verify(unsigned int* sOut, unsigned int* cudaOut) {

	for (int i = 0; i < HISTOBINS; i++) {
		if (sOut[i] != cudaOut[i]) {
			fprintf(stderr, "verify failed! cOut[%d]=%d cudaOut[%d]=%d \n", i, sOut[i], i, cudaOut[i]);
			return 0;
		}
	}
	return 1;
}


int main()
{
	int xSize, ySize;
	xSize = XSIZE; ySize = YSIZE;
	unsigned int* histoCTable, * histoCudaTableAtomic, * histoCudaTable;
	unsigned int* histoCudaSharedTableSync, * histoCudaSharedMemBlock;
	unsigned int* in;
	in = new unsigned int[xSize * ySize];
	histoCTable = new unsigned int[HISTOBINS];
	histoCudaTable = new unsigned int[HISTOBINS];
	histoCudaTableAtomic = new unsigned int[HISTOBINS];
	histoCudaSharedMemBlock = new unsigned int[HISTOBINS];
	int histoBins = HISTOBINS;

	float dCpuTime;
	int loopCount;
	CPerfCounter counter;

	for (int i = 0; i < ySize; i++)
		for (int j = 0; j < xSize; j++)
			in[i * xSize + j] = (i * xSize + j) % 256;

	for (int i = 0; i < HISTOBINS; i++) {
		histoCTable[i] = 0;
		histoCudaTable[i] = 0;
		histoCudaTableAtomic[i] = 0;
		histoCudaSharedMemBlock[i] = 0;
	}

	dCpuTime = 0.0f;
	int verifyResult;

	dCpuTime = 0.0f;
	for (loopCount = 0; loopCount < 100; loopCount++) {
		for (int i = 0; i < HISTOBINS; i++) {
			histoCTable[i] = 0;
		}
		counter.Reset();
		counter.Start();
		histoWithC(in, histoCTable, xSize, ySize);
		counter.Stop();
		dCpuTime += counter.GetElapsedTime();
		
	}

	//dCpuTime = counter.GetElapsedTime()/(double)loopCount;
	printf("Histo C Performance (ms) = %f \n", dCpuTime / (double)loopCount * 1000.0);
	

	// square vectors in parallel.

	hipError_t cudaStatus = histoWithCuda(in, histoCudaTable, xSize, ySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaHistogram failed! \n");
		return 1;
	}
	verifyResult = verify(histoCTable, histoCudaTable);
	if (verifyResult == 0) {
		fprintf(stderr, "Verify histoCudaTable Failed \n");
	}
	else {
		fprintf(stderr, "Verify histoCudaTable Successfule \n");
	}


	cudaStatus = histoWithCudaAtomic(in, histoCudaTableAtomic, xSize, ySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaHistogram failed! \n");
		return 1;
	}
	verifyResult = verify(histoCTable, histoCudaTableAtomic);
	if (verifyResult == 0) {
		fprintf(stderr, "Verify histoCudaTableAtomic Failed \n");
	}
	else {
		fprintf(stderr, "Verify histoCudaTableAtomic Successfule \n");
	}


	cudaStatus = histoSharedMem(in, histoCudaSharedMemBlock, xSize, ySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaSharedMemHistogram failed! \n");
		return 1;
	}

	verifyResult = verify(histoCTable, histoCudaSharedMemBlock);
	if (verifyResult == 0) {
		fprintf(stderr, "Verify histoSharedMem Failed \n");
	}
	else {
		fprintf(stderr, "Verify histoSharedMem Successfule \n");
	}


	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed! \n");
		return 1;
	}



	printf("index\t\tCTable\t\tCudaTable\tCudaTableAtomic\tCudaSharedTable\n");
	for (int i = 0; i < HISTOBINS; i++) {
		printf("%d\t\t%d\t\t%d\t\t\t%d\t\t%d\t\t%d \n", i, histoCTable[i], histoCudaTable[i], histoCudaTableAtomic[i], histoCudaSharedTableSync[i], histoCudaSharedMemBlock[i]);
	}

	delete[] histoCudaTable;
	delete[] histoCTable;
	delete[] in;

	return 0;
}

//  function for using CUDA to square vectors in parallel.
hipError_t histoWithCuda(unsigned int* hIn, unsigned int* hOut, int xSize, int ySize)
{
	unsigned int* dev_in = 0;
	unsigned int* dev_out = 0;
	GpuTimer timer;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	hipEventRecord(start, 0); // To measure performance

	cudaStatus = hipMalloc((void**)&dev_in, xSize * ySize * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_out, HISTOBINS * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	hipMemset((void*)&dev_out, 0, HISTOBINS * sizeof(int));

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_in, hIn, xSize * ySize * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	dim3 numThreads(32, 32);
	dim3 numBlocks((xSize + numThreads.x - 1) / numThreads.x, (ySize + numThreads.y - 1) / numThreads.y);

	timer.Start();
	histoKernel << <numBlocks, numThreads >> > (dev_in, dev_out, xSize * ySize);
	cudaStatus = hipDeviceSynchronize();
	timer.Stop();
	printf("With histoKernel Time  elapsed=%g ms\n", timer.Elapsed());

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}
	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(hOut, dev_out, HISTOBINS * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	//printf("Time elapsed=%g ms\n", timer.Elapsed());

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	float cudaElapsedTime;
	//hipEventElapsedTime(&cudaElapsedTime, start, stop);
	//printf("Time for execution = %3.1f ms \n", cudaElapsedTime); 
	hipEventDestroy(start);
	hipEventDestroy(stop);

Error:
	hipFree(dev_in);
	hipFree(dev_out);

	return cudaStatus;
}

//  function for using CUDA to square vectors in parallel.
hipError_t histoWithCudaAtomic(unsigned int* hIn, unsigned int* hOut, int xSize, int ySize)
{
	unsigned int* dev_in = 0;
	unsigned int* dev_out = 0;
	GpuTimer timer;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	hipEventRecord(start, 0); // To measure performance

	cudaStatus = hipMalloc((void**)&dev_in, xSize * ySize * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_out, HISTOBINS * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	hipMemset((void*)&dev_out, 0, HISTOBINS * sizeof(int));

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_in, hIn, xSize * ySize * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	dim3 numThreads(32, 32);
	dim3 numBlocks((xSize + numThreads.x - 1) / numThreads.x, (ySize + numThreads.y - 1) / numThreads.y);

	timer.Start();
	histoAtomicKernel << <numBlocks, numThreads >> > (dev_in, dev_out, xSize * ySize);
	cudaStatus = hipDeviceSynchronize();
	timer.Stop();
	printf("With histoAtomicKernel Time  elapsed=%g ms\n", timer.Elapsed());


	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}
	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(hOut, dev_out, HISTOBINS * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	//printf("Time elapsed=%g ms\n", timer.Elapsed());

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	float cudaElapsedTime;
	//hipEventElapsedTime(&cudaElapsedTime, start, stop);
	//printf("Time for execution = %3.1f ms \n", cudaElapsedTime); 
	hipEventDestroy(start);
	hipEventDestroy(stop);

Error:
	hipFree(dev_in);
	hipFree(dev_out);

	return cudaStatus;
}


hipError_t histoSharedMem(unsigned int* hIn, unsigned int* hOut, int xSize, int ySize)
{
	//__shared__ sharedMem[];
	unsigned int* dev_in = 0;
	unsigned int* dev_out = 0;
	unsigned int* dev_histoBlock_out = 0;
	GpuTimer timer;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}


	dim3 numThreads(32, 32);
	dim3 numBlocks((xSize + numThreads.x - 1) / numThreads.x, (ySize + numThreads.y - 1) / numThreads.y);

	hipEventRecord(start, 0); // To measure performance

	//To store intermediate results from blocks
	cudaStatus = hipMalloc((void**)&dev_histoBlock_out, numBlocks.x * numBlocks.y* HISTOBINS * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	hipMemset((void*)&dev_histoBlock_out, 0, HISTOBINS * numBlocks.x * numBlocks.y* sizeof(int));

	cudaStatus = hipMalloc((void**)&dev_in, xSize * ySize * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_out, HISTOBINS * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	hipMemset((void*)&dev_out, 0, HISTOBINS * sizeof(int));

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_in, hIn, xSize * ySize * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	timer.Start();
	histoSharedKernel << <numBlocks, numThreads >> > (dev_in, dev_histoBlock_out, xSize, ySize);
	histoAccm<< <1, HISTOBINS >> > (dev_histoBlock_out, dev_out, numBlocks.x * numBlocks.y* HISTOBINS);
	cudaStatus = hipDeviceSynchronize();
	timer.Stop();
	printf("With histoSharedKernel  Time  elapsed=%g ms\n", timer.Elapsed());

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}
	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(hOut, dev_out, HISTOBINS * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	//printf("Time elapsed=%g ms\n", timer.Elapsed());

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);


	hipEventDestroy(start);
	hipEventDestroy(stop);

Error:
	hipFree(dev_in);
	hipFree(dev_out);

	return cudaStatus;
}
