#include "hip/hip_runtime.h"
/*
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"

#include <stdio.h>

#define N (1024*1024)
#define FULL_DATA_SIZE (N*20)
#define CHUNK_SIZE (N/3) // Smaller chunk size to better utilize streams

__global__ void kernel(int* a, int* b, int* c, int dataSize) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < dataSize) {
        int idx1 = (idx + 1) % 256;
        int idx2 = (idx + 2) % 256;
        float as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
        float bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;
        c[idx] = (as + bs) / 2;
    }
}

int main(void) {
    hipDeviceProp_t prop;
    int whichDevice;
    hipGetDevice(&whichDevice);
    hipGetDeviceProperties(&prop, whichDevice);
    if (!prop.deviceOverlap) {
        printf("Device will not handle overlaps, so no speed up from streams\n");
        return 0;
    }

    hipEvent_t start, stop;
    float elapsedTime;

    hipStream_t stream0, stream1, stream2;
    int* host_a, * host_b, * host_c;
    int* dev_a0, * dev_b0, * dev_c0;
    int* dev_a1, * dev_b1, * dev_c1;
    int* dev_a2, * dev_b2, * dev_c2;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipStreamCreate(&stream0);
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    hipMalloc((void**)&dev_a0, CHUNK_SIZE * sizeof(int));
    hipMalloc((void**)&dev_b0, CHUNK_SIZE * sizeof(int));
    hipMalloc((void**)&dev_c0, CHUNK_SIZE * sizeof(int));

    hipMalloc((void**)&dev_a1, CHUNK_SIZE * sizeof(int));
    hipMalloc((void**)&dev_b1, CHUNK_SIZE * sizeof(int));
    hipMalloc((void**)&dev_c1, CHUNK_SIZE * sizeof(int));

    hipMalloc((void**)&dev_a2, CHUNK_SIZE * sizeof(int));
    hipMalloc((void**)&dev_b2, CHUNK_SIZE * sizeof(int));
    hipMalloc((void**)&dev_c2, CHUNK_SIZE * sizeof(int));

    hipHostAlloc((void**)&host_a, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**)&host_b, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**)&host_c, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);

    for (int i = 0; i < FULL_DATA_SIZE; i++) {
        host_a[i] = rand();
        host_b[i] = rand();
    }

    hipEventRecord(start, 0);

    for (int i = 0; i < FULL_DATA_SIZE; i += CHUNK_SIZE * 3) {
        // 1st stream
        hipMemcpyAsync(dev_a0, host_a + i, CHUNK_SIZE * sizeof(int), hipMemcpyHostToDevice, stream0);
        hipMemcpyAsync(dev_b0, host_b + i, CHUNK_SIZE * sizeof(int), hipMemcpyHostToDevice, stream0);
        kernel << <CHUNK_SIZE / 256, 256, 0, stream0 >> > (dev_a0, dev_b0, dev_c0, CHUNK_SIZE);
        hipMemcpyAsync(host_c + i, dev_c0, CHUNK_SIZE * sizeof(int), hipMemcpyDeviceToHost, stream0);

        // 2nd stream
        if (i + CHUNK_SIZE < FULL_DATA_SIZE) {
            hipMemcpyAsync(dev_a1, host_a + i + CHUNK_SIZE, CHUNK_SIZE * sizeof(int), hipMemcpyHostToDevice, stream1);
            hipMemcpyAsync(dev_b1, host_b + i + CHUNK_SIZE, CHUNK_SIZE * sizeof(int), hipMemcpyHostToDevice, stream1);
            kernel << <CHUNK_SIZE / 256, 256, 0, stream1 >> > (dev_a1, dev_b1, dev_c1, CHUNK_SIZE);
            hipMemcpyAsync(host_c + i + CHUNK_SIZE, dev_c1, CHUNK_SIZE * sizeof(int), hipMemcpyDeviceToHost, stream1);
        }

        // 3rd stream
        if (i + 2 * CHUNK_SIZE < FULL_DATA_SIZE) {
            hipMemcpyAsync(dev_a2, host_a + i + 2 * CHUNK_SIZE, CHUNK_SIZE * sizeof(int), hipMemcpyHostToDevice, stream2);
            hipMemcpyAsync(dev_b2, host_b + i + 2 * CHUNK_SIZE, CHUNK_SIZE * sizeof(int), hipMemcpyHostToDevice, stream2);
            kernel << <CHUNK_SIZE / 256, 256, 0, stream2 >> > (dev_a2, dev_b2, dev_c2, CHUNK_SIZE);
            hipMemcpyAsync(host_c + i + 2 * CHUNK_SIZE, dev_c2, CHUNK_SIZE * sizeof(int), hipMemcpyDeviceToHost, stream2);
        }
    }

    hipStreamSynchronize(stream0);
    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Time taken:  %3.1f ms\n", elapsedTime);

    hipHostFree(host_a);
    hipHostFree(host_b);
    hipHostFree(host_c);
    hipFree(dev_a0);
    hipFree(dev_b0);
    hipFree(dev_c0);
    hipFree(dev_a1);
    hipFree(dev_b1);
    hipFree(dev_c1);
    hipFree(dev_a2);
    hipFree(dev_b2);
    hipFree(dev_c2);
    hipStreamDestroy(stream0);
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);

    return 0;
}
*/
/*
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"

#include <stdio.h>

//#define SINGLE_STREAM
//#define DOUBLE_STREAM

#define N   (1024*1024)
#define FULL_DATA_SIZE   (N*20)


__global__ void kernel(int* a, int* b, int* c) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        int idx1 = (idx + 1) % 256;
        int idx2 = (idx + 2) % 256;
        float   as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
        float   bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;
        c[idx] = (as + bs) / 2;
    }
}


int main(void) {

    hipDeviceProp_t  prop;
    int whichDevice;
    (hipGetDevice(&whichDevice));
    (hipGetDeviceProperties(&prop, whichDevice));
    if (!prop.deviceOverlap) {
        printf("Device will not handle overlaps, so no speed up from streams\n");
        return 0;
    }

    hipEvent_t     start, stop;
    float           elapsedTime;

    hipStream_t    stream0, stream1;// , stream2;
    int* host_a, * host_b, * host_c;

    //  GPU beffers for stream N
    int* dev_a0, * dev_b0, * dev_c0; // buffers
    int* dev_a1, * dev_b1, * dev_c1;
    //int* dev_a2, * dev_b2, * dev_c2;

    // start the timers
    (hipEventCreate(&start));
    (hipEventCreate(&stop));

    // initialize the stream
    (hipStreamCreate(&stream0));
    (hipStreamCreate(&stream1));
    //(hipStreamCreate(&stream2));

    // allocate the memory on the GPU
    (hipMalloc((void**)&dev_a0,
        N * sizeof(int)));
    (hipMalloc((void**)&dev_b0,
        N * sizeof(int)));
    (hipMalloc((void**)&dev_c0,
        N * sizeof(int)));

    (hipMalloc((void**)&dev_a1,
        N * sizeof(int)));
    (hipMalloc((void**)&dev_b1,
        N * sizeof(int)));
    (hipMalloc((void**)&dev_c1,
        N * sizeof(int)));

    //(hipMalloc((void**)&dev_a2,
    //    N * sizeof(int)));
    //(hipMalloc((void**)&dev_b2,
    //    N * sizeof(int)));
    //(hipMalloc((void**)&dev_c2,
    //    N * sizeof(int)));

    // allocate host locked memory, used to stream
    (hipHostAlloc((void**)&host_a,
        FULL_DATA_SIZE * sizeof(int),
        hipHostMallocDefault));
    (hipHostAlloc((void**)&host_b,
        FULL_DATA_SIZE * sizeof(int),
        hipHostMallocDefault));
    (hipHostAlloc((void**)&host_c,
        FULL_DATA_SIZE * sizeof(int),
        hipHostMallocDefault));

    for (int i = 0; i < FULL_DATA_SIZE; i++) {
        host_a[i] = rand();
        host_b[i] = rand();
    }

    (hipEventRecord(start, 0));
    // now loop over full data, in bite-sized chunks
    for (int i = 0; i < FULL_DATA_SIZE; i += N * 2) {
        //1st work
        // copy the locked memory to the device, async
        (hipMemcpyAsync(dev_a0, host_a + i,
            N * sizeof(int),
            hipMemcpyHostToDevice,
            stream0));
        (hipMemcpyAsync(dev_a1, host_a + i + N,
            N * sizeof(int),
            hipMemcpyHostToDevice,
            stream1));

        //(hipMemcpyAsync(dev_a2, host_a + i + N*2,
        //    N * sizeof(int),
        //    hipMemcpyHostToDevice,
        //    stream2));


        (hipMemcpyAsync(dev_b0, host_b + i,
            N * sizeof(int),
            hipMemcpyHostToDevice,
            stream0));
        (hipMemcpyAsync(dev_b1, host_b + i + N,
            N * sizeof(int),
            hipMemcpyHostToDevice,
            stream1));
        //(hipMemcpyAsync(dev_b2, host_b + i + N*2,
        //    N * sizeof(int),
        //    hipMemcpyHostToDevice,
        //    stream2));

        kernel << <N / 256, 256, 0, stream0 >> > (dev_a0, dev_b0, dev_c0);

        kernel << <N / 256, 256, 0, stream1 >> > (dev_a1, dev_b1, dev_c1);

        //kernel << <N / 256, 256, 0, stream2 >> > (dev_a2, dev_b2, dev_c2);

        // copy the data from device to locked memory
        (hipMemcpyAsync(host_c + i, dev_c0,
            N * sizeof(int),
            hipMemcpyDeviceToHost,
            stream0));


        // copy the data from device to locked memory
        (hipMemcpyAsync(host_c + i + N, dev_c1,
            N * sizeof(int),
            hipMemcpyDeviceToHost,
            stream1));

        //(hipMemcpyAsync(host_c + i + N, dev_c2,
        //    N * sizeof(int),
        //    hipMemcpyDeviceToHost,
        //    stream2));

    }
    // copy result chunk from locked to full buffer
    (hipStreamSynchronize(stream0));
    (hipStreamSynchronize(stream1));
    //(hipStreamSynchronize(stream2));

    (hipEventRecord(stop, 0));

    (hipEventSynchronize(stop));
    (hipEventElapsedTime(&elapsedTime,
        start, stop));
    printf("Time taken:  %3.1f ms\n", elapsedTime);

    // cleanup the streams and memory
    (hipHostFree(host_a));
    (hipHostFree(host_b));
    (hipHostFree(host_c));
    (hipFree(dev_a0));
    (hipFree(dev_b0));
    (hipFree(dev_c0));
    (hipFree(dev_a1));
    (hipFree(dev_b1));
    (hipFree(dev_c1));
    //(hipFree(dev_a2));
    //(hipFree(dev_b2));
    //(hipFree(dev_c2));
    (hipStreamDestroy(stream0));
    (hipStreamDestroy(stream1));
    //(hipStreamDestroy(stream2));

    return 0;

}

*/

/*

#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"

#include <stdio.h>

#define N   (1024*1024)
#define FULL_DATA_SIZE   (N*20)

__global__ void kernel(int* a, int* b, int* c) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        int idx1 = (idx + 1) % 256;
        int idx2 = (idx + 2) % 256;
        float   as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
        float   bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;
        c[idx] = (as + bs) / 2;
    }
}

int main(void) {

    hipDeviceProp_t  prop;
    int whichDevice;
    hipGetDevice(&whichDevice);
    hipGetDeviceProperties(&prop, whichDevice);
    if (!prop.deviceOverlap) {
        printf("Device will not handle overlaps, so no speed up from streams\n");
        return 0;
    }

    hipEvent_t     start, stop;
    float           elapsedTime;

    hipStream_t    stream0, stream1, stream2, stream3;
    int* host_a, * host_b, * host_c;

    //  GPU buffers for streams
    int* dev_a0, * dev_b0, * dev_c0;
    int* dev_a1, * dev_b1, * dev_c1;
    int* dev_a2, * dev_b2, * dev_c2;
    int* dev_a3, * dev_b3, * dev_c3;

    // start the timers
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // initialize the streams
    hipStreamCreate(&stream0);
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    hipStreamCreate(&stream3);

    // allocate the memory on the GPU
    hipMalloc((void**)&dev_a0, N * sizeof(int));
    hipMalloc((void**)&dev_b0, N * sizeof(int));
    hipMalloc((void**)&dev_c0, N * sizeof(int));

    hipMalloc((void**)&dev_a1, N * sizeof(int));
    hipMalloc((void**)&dev_b1, N * sizeof(int));
    hipMalloc((void**)&dev_c1, N * sizeof(int));

    hipMalloc((void**)&dev_a2, N * sizeof(int));
    hipMalloc((void**)&dev_b2, N * sizeof(int));
    hipMalloc((void**)&dev_c2, N * sizeof(int));

    hipMalloc((void**)&dev_a3, N * sizeof(int));
    hipMalloc((void**)&dev_b3, N * sizeof(int));
    hipMalloc((void**)&dev_c3, N * sizeof(int));

    // allocate host locked memory, used to stream
    hipHostAlloc((void**)&host_a, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**)&host_b, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**)&host_c, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);

    for (int i = 0; i < FULL_DATA_SIZE; i++) {
        host_a[i] = rand();
        host_b[i] = rand();
    }

    hipEventRecord(start, 0);
    // now loop over full data, in bite-sized chunks
    for (int i = 0; i < FULL_DATA_SIZE; i += N) {
        int stream_idx = (i / N) % 4;

        hipStream_t current_stream;
        int* dev_a;
        int* dev_b;
        int* dev_c;

        if (stream_idx == 0) {
            current_stream = stream0;
            dev_a = dev_a0;
            dev_b = dev_b0;
            dev_c = dev_c0;
        }
        else if (stream_idx == 1) {
            current_stream = stream1;
            dev_a = dev_a1;
            dev_b = dev_b1;
            dev_c = dev_c1;
        }
        else if (stream_idx == 2) {
            current_stream = stream2;
            dev_a = dev_a2;
            dev_b = dev_b2;
            dev_c = dev_c2;
        }
        else {
            current_stream = stream3;
            dev_a = dev_a3;
            dev_b = dev_b3;
            dev_c = dev_c3;
        }

        // copy the locked memory to the device, async
        hipMemcpyAsync(dev_a, host_a + i, N * sizeof(int), hipMemcpyHostToDevice, current_stream);
        hipMemcpyAsync(dev_b, host_b + i, N * sizeof(int), hipMemcpyHostToDevice, current_stream);

        kernel << <N / 256, 256, 0, current_stream >> > (dev_a, dev_b, dev_c);

        // copy the data from device to locked memory
        hipMemcpyAsync(host_c + i, dev_c, N * sizeof(int), hipMemcpyDeviceToHost, current_stream);
    }

    // synchronize the streams
    hipStreamSynchronize(stream0);
    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);
    hipStreamSynchronize(stream3);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Time taken:  %3.1f ms\n", elapsedTime);

    // cleanup the streams and memory
    hipHostFree(host_a);
    hipHostFree(host_b);
    hipHostFree(host_c);
    hipFree(dev_a0);
    hipFree(dev_b0);
    hipFree(dev_c0);
    hipFree(dev_a1);
    hipFree(dev_b1);
    hipFree(dev_c1);
    hipFree(dev_a2);
    hipFree(dev_b2);
    hipFree(dev_c2);
    hipFree(dev_a3);
    hipFree(dev_b3);
    hipFree(dev_c3);
    hipStreamDestroy(stream0);
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    hipStreamDestroy(stream3);

    return 0;
}


*/


#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"

#include <stdio.h>

#define N   (1024*1024)
#define FULL_DATA_SIZE   (N*20)
#define NUM_STREAMS 2
#define CHUNK_SIZE (N / NUM_STREAMS)

__global__ void kernel(int* a, int* b, int* c) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < CHUNK_SIZE) {
        int idx1 = (idx + 1) % 256;
        int idx2 = (idx + 2) % 256;
        float   as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
        float   bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;
        c[idx] = (as + bs) / 2;
    }
}

int main(void) {

    hipDeviceProp_t  prop;
    int whichDevice;
    hipGetDevice(&whichDevice);
    hipGetDeviceProperties(&prop, whichDevice);
    if (!prop.deviceOverlap) {
        printf("Device will not handle overlaps, so no speed up from streams\n");
        return 0;
    }

    hipEvent_t     start, stop;
    float           elapsedTime;

    hipStream_t    streams[NUM_STREAMS];
    int* host_a, * host_b, * host_c;

    //  GPU buffers for streams
    int* dev_a[NUM_STREAMS], * dev_b[NUM_STREAMS], * dev_c[NUM_STREAMS];

    // start the timers
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // initialize the streams and allocate the memory on the GPU
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamCreate(&streams[i]);
        hipMalloc((void**)&dev_a[i], CHUNK_SIZE * sizeof(int));
        hipMalloc((void**)&dev_b[i], CHUNK_SIZE * sizeof(int));
        hipMalloc((void**)&dev_c[i], CHUNK_SIZE * sizeof(int));
    }

    // allocate host locked memory, used to stream
    hipHostAlloc((void**)&host_a, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**)&host_b, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**)&host_c, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);

    for (int i = 0; i < FULL_DATA_SIZE; i++) {
        host_a[i] = rand();
        host_b[i] = rand();
    }

    hipEventRecord(start, 0);
    // now loop over full data, in bite-sized chunks
    for (int i = 0; i < FULL_DATA_SIZE; i += N * 2)
    {
        int idx = (i / N) % NUM_STREAMS;
        (hipMemcpyAsync(dev_a[idx], host_a + i,
            CHUNK_SIZE * sizeof(int),
            hipMemcpyHostToDevice,
            streams[idx]));
        (hipMemcpyAsync(dev_b[idx], host_b + i,
            CHUNK_SIZE * sizeof(int),
            hipMemcpyHostToDevice,
            streams[idx]));

        kernel << <CHUNK_SIZE / 256, 256, 0, streams[idx] >> > (dev_a[idx], dev_b[idx], dev_c[idx]);

        hipMemcpyAsync(host_c + i, dev_c[idx], CHUNK_SIZE * sizeof(int), hipMemcpyDeviceToHost, streams[idx]);
    }

    // synchronize the streams
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamSynchronize(streams[i]);
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Time taken:  %3.1f ms\n", elapsedTime);

    printf("Number of Streams: %d\n", NUM_STREAMS);
    printf("Full Data Size: %d\n", FULL_DATA_SIZE);
    printf("Data Chunk Size (FULL_DATA_SIZE/(20 * STREAM_NUM)): %d\n", CHUNK_SIZE);
    printf("Time taken:  %3.1f ms\n", elapsedTime);

    // cleanup the streams and memory
    hipHostFree(host_a);
    hipHostFree(host_b);
    hipHostFree(host_c);
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipFree(dev_a[i]);
        hipFree(dev_b[i]);
        hipFree(dev_c[i]);
        hipStreamDestroy(streams[i]);
    }

    return 0;
}



