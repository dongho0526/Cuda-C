#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"

#include <stdio.h>

#define thread_num 1024
#define block_num 1
#define block_num_x 1
#define block_num_y 512

hipError_t Add(unsigned char *input, unsigned char *output, int xSize, int ySize);
hipError_t FlipX(unsigned char* input, unsigned char* output, int xSize, int ySize);
hipError_t FlipY(unsigned char* input, unsigned char* output, int xSize, int ySize);


__global__ void AddKernel_1thread(unsigned char* input, unsigned char* output, int xSize, int ySize)
{

    int idx = threadIdx.x; // thread => 0
    
    // Ensure that we only have one thread and one block
    if (idx == 0) {
        for (int i = 0; i < xSize * ySize; i++) {
            int temp = input[i] + input[i];
            output[i] = (temp > 255) ? 255 : temp;
        }
    }

}

__global__ void FlipYKernel_1thread(unsigned char* input, unsigned char* output, int xSize, int ySize) {
    int idx = threadIdx.x;
    int idy = threadIdx.y;

    if (idx == 0 && idy == 0) {
        for (int y = 0; y < ySize; ++y) {
            for (int x = 0; x < xSize; ++x) {
                int inputIndex = y * xSize + x;
                int outputIndex = y * xSize + (xSize - 1 - x);
                output[outputIndex] = input[inputIndex];
                
            }
        }
    }

}

__global__ void FlipXKernel_1thread(unsigned char* input, unsigned char* output, int xSize, int ySize) {
    int idx = threadIdx.x;
    int idy = threadIdx.y;

    if (idx == 0 && idy == 0) {
        for (int y = 0; y < ySize; ++y) {
            for (int x = 0; x < xSize; ++x) {
                int inputIndex = y * xSize + x;
                int outputIndex = (ySize - y - 1) * xSize + x;
                output[outputIndex] = input[inputIndex];
            }
        }
    }
}


__global__ void AddKernel_1024(unsigned char* input, unsigned char* output, int xSize, int ySize)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // 각 스레드의 고유 인덱스
    int xySize = xSize * ySize;
    int numThreads = blockDim.x * gridDim.x; // 전체 스레드 수

    for (int i = idx; i < xySize; i += numThreads) {
        int temp = input[i] + input[i];
        output[i] = (temp > 255) ? 255 : temp;
    }
}

__global__ void FlipXKernel_1024(unsigned char* input, unsigned char* output, int xSize, int ySize)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int xySize = xSize * ySize;
    int numThreadsx = blockDim.x * gridDim.x;

    for (int i = idx; i < xySize; i += numThreadsx) {
        int x = i % xSize;
        int y = i / xSize;
        int inputIndex = y * xSize + x;
        int outputIndex = (ySize - 1 - y) * xSize + x;
        output[outputIndex] = input[inputIndex];
    }

}

__global__ void FlipYKernel_1024(unsigned char* input, unsigned char* output, int xSize, int ySize)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int xySize = xSize * ySize;
    int numThreadsx = blockDim.x * gridDim.x;

    for (int i = idx; i < xySize; i += numThreadsx) {
        int x = i % xSize;
        int y = i / xSize;
        int inputIndex = y * xSize + x;
        int outputIndex = y * xSize + (xSize - 1 - x);
        output[outputIndex] = input[inputIndex];
    }
}

__global__ void AddKernel_512x512(unsigned char* input, unsigned char* output, int xSize, int ySize)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < xSize && y < ySize) {
        int idx = y * xSize + x;
        int temp = input[idx] + input[idx];
        output[idx] = (temp > 255) ? 255 : temp;
    }
}

__global__ void FlipXKernel_512x512(unsigned char* input, unsigned char* output, int xSize, int ySize)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < xSize && y < ySize) {
        int inputIndex = y * xSize + x;
        int outputIndex = (ySize-1-y) * xSize + x;
        output[outputIndex] = input[inputIndex];
    }
}

__global__ void FlipYKernel_512x512(unsigned char* input, unsigned char* output, int xSize, int ySize)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < xSize && y < ySize) {
        int inputIndex = y * xSize + x;
        int outputIndex = y * xSize + (xSize - 1 - x);
        output[outputIndex] = input[inputIndex];
    }

}

// Zblock은 blockSize => (xSize, 1) gridSize => (ySize, 1)

__global__ void AddKernel_Zblock(unsigned char* input, unsigned char* output, int xSize, int ySize)
{
    int x = threadIdx.x;
    int y = blockIdx.x;

    if (x < xSize && y < ySize) {
        int idx = y * xSize + x;
        int temp = input[idx] + input[idx];
        output[idx] = (temp > 255) ? 255 : temp;
    }

}

__global__ void FlipXKernel_Zblock(unsigned char* input, unsigned char* output, int xSize, int ySize)
{
    int x = threadIdx.x;
    int y = blockIdx.x;

    if (x < xSize && y < ySize) {
        int inputIndex = y * xSize + x;
        int outputIndex = (ySize - 1 - y) * xSize + x;
        output[outputIndex] = input[inputIndex];
    }
}

__global__ void FlipYKernel_Zblock(unsigned char* input, unsigned char* output, int xSize, int ySize)
{
    int x = threadIdx.x;
    int y = blockIdx.x;

    if (x < xSize && y < ySize) {
        int inputIndex = y * xSize + x;
        int outputIndex = y * xSize + (xSize - 1 - x);
        output[outputIndex] = input[inputIndex];
    }

}

// Mblock은 blockSize => (xSize/blocksize, ySize/blocksize) gridSize => (16, 16)

__global__ void AddKernel_Mblock(unsigned char* input, unsigned char* output, int xSize, int ySize)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < xSize && y < ySize) {
        int idx = y * xSize + x;
        int temp = input[idx] + input[idx];
        output[idx] = (temp > 255) ? 255 : temp;
    }
}

__global__ void FlipXKernel_Mblock(unsigned char* input, unsigned char* output, int xSize, int ySize)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < xSize && y < ySize) {
        int inputIndex = y * xSize + x;
        int outputIndex = (ySize - 1 - y) * xSize + x;
        output[outputIndex] = input[inputIndex];
    }
}

__global__ void FlipYKernel_Mblock(unsigned char* input, unsigned char* output, int xSize, int ySize)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < xSize && y < ySize) {
        int inputIndex = y * xSize + x;
        int outputIndex = y * xSize + (xSize - 1 - x);
        output[outputIndex] = input[inputIndex];
    }

}


int main()
{
    unsigned char* input, * CudaOutput, * CudaOutput_X, * CudaOutput_Y;
    int xSize, ySize;
    
    xSize = 512;
    ySize = 512;

    input = new unsigned char[xSize * ySize];
    CudaOutput = new unsigned char[xSize * ySize];
    CudaOutput_X = new unsigned char[xSize * ySize];
    CudaOutput_Y = new unsigned char[xSize * ySize];

    int i, j;

    FILE* fp;
    fp = fopen("usc.raw", "rb");
    fread(input, xSize, ySize, fp);

    //Add(input, CudaOutput, xSize, ySize);
    
    hipError_t cudaStatus = Add(input, CudaOutput, xSize, ySize);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Add failed!\n");
        return 1;
    }

    fp = fopen("Output_add.raw", "wb");
    fwrite(CudaOutput, xSize, ySize, fp);
    fclose(fp);

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    
    hipError_t cudaStatus_X = FlipX(input, CudaOutput_X, xSize, ySize);

    if (cudaStatus_X != hipSuccess) {
        fprintf(stderr, "FlipX failed\n");
        return 1;
    }

    fp = fopen("Output_FlipX.raw", "wb");
    fwrite(CudaOutput_X, xSize, ySize, fp);
    fclose(fp);

    cudaStatus_X = hipDeviceReset();
    if (cudaStatus_X != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }


    hipError_t cudaStatus_Y = FlipY(input, CudaOutput_Y, xSize, ySize);

    if (cudaStatus_Y != hipSuccess) {
        fprintf(stderr, "FlipY failed\n");
        return 1;
    }

    fp = fopen("Output_FlipY.raw", "wb");
    fwrite(CudaOutput_Y, xSize, ySize, fp);
    fclose(fp);

    cudaStatus_Y = hipDeviceReset();
    if (cudaStatus_Y != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }



    delete[] CudaOutput;
    delete[] CudaOutput_X;
    delete[] CudaOutput_Y;
    delete[] input;

    return 0;
}

hipError_t Add(unsigned char* input, unsigned char* output, int xSize, int ySize)
{
    unsigned char* dev_input = 0;
    unsigned char* dev_output = 0;

    unsigned int xySize = xSize * ySize;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipError_t cudaStatus;

    hipDeviceProp_t prop;
    int count;

    hipGetDeviceCount(&count);

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    }


    hipEventRecord(start, 0);

    cudaStatus = hipMalloc((void**)&dev_input, xySize * sizeof(char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        //goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_output, xySize * sizeof(char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        //goto Error;
    }

    cudaStatus = hipMemcpy(dev_input, input, xySize, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        //goto Error;
    }

    dim3 blockSize(16, 16); // 1D 블록, 1024 스레드
    dim3 gridSize(32, 32); // 1D 그리드, 1 블록

    //AddKernel_1thread << <1, 1 >> > (dev_input, dev_output, xSize, ySize);
    //AddKernel << < blockSize, gridSize >> > (dev_input, dev_output, xSize, ySize);
    //AddKernel_512x512 << < blockSize, gridSize >> > (dev_input, dev_output, xSize, ySize);
    //AddKernel_Zblock << <blockSize, gridSize >> > (dev_input, dev_output, xSize, ySize);
    AddKernel_Mblock << <blockSize, gridSize >> > (dev_input, dev_output, xSize, ySize);

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching invert8Kernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(output, dev_output, xySize * sizeof(char), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }


    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float cudaElapsedTime;
    hipEventElapsedTime(&cudaElapsedTime, start, stop);
    printf("Add - Time for execution = %3.1f ms \n", cudaElapsedTime);
    hipEventDestroy(start);
    hipEventDestroy(stop);

Error:
    hipFree(dev_input);
    hipFree(dev_output);

    return cudaStatus;
}




hipError_t FlipY(unsigned char* input, unsigned char* output, int xSize, int ySize)
{
    unsigned char* dev_input = 0;
    unsigned char* dev_output = 0;

    unsigned int xySize = xSize * ySize;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipError_t cudaStatus_X;

    hipDeviceProp_t prop;
    int count;

    hipGetDeviceCount(&count);

    cudaStatus_X = hipSetDevice(0);
    if (cudaStatus_X != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    }


    hipEventRecord(start, 0);

    cudaStatus_X = hipMalloc((void**)&dev_input, xySize * sizeof(char));
    if (cudaStatus_X != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        //goto Error;
    }

    cudaStatus_X = hipMalloc((void**)&dev_output, xySize * sizeof(char));
    if (cudaStatus_X != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        //goto Error;
    }

    cudaStatus_X = hipMemcpy(dev_input, input, xySize, hipMemcpyHostToDevice);
    if (cudaStatus_X != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        //goto Error;
    }

    dim3 blockSize(16, 16); // 1D 블록, 1024 스레드
    dim3 gridSize(32, 32); // 1D 그리드, 1 블록

    //FlipYKernel << <blockSize, gridSize >> > (dev_input, dev_output, xSize, ySize);
    //FlipYKernel_1024 << <blockSize, gridSize >> > (dev_input, dev_output, xSize, ySize);
    //FlipYKernel_512x512 << <blockSize, gridSize >> > (dev_input, dev_output, xSize, ySize);
    //FlipYKernel_Zblock << <blockSize, gridSize >> > (dev_input, dev_output, xSize, ySize);
    FlipYKernel_Mblock << <blockSize, gridSize >> > (dev_input, dev_output, xSize, ySize);

    cudaStatus_X = hipDeviceSynchronize();
    if (cudaStatus_X != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching invert8Kernel!\n", cudaStatus_X);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus_X = hipMemcpy(output, dev_output, xySize * sizeof(char), hipMemcpyDeviceToHost);
    if (cudaStatus_X != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }


    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float cudaElapsedTime;
    hipEventElapsedTime(&cudaElapsedTime, start, stop);
    printf("FlipY - Time for execution = %3.1f ms \n", cudaElapsedTime);
    hipEventDestroy(start);
    hipEventDestroy(stop);

Error:
    hipFree(dev_input);
    hipFree(dev_output);

    return cudaStatus_X;
}


hipError_t FlipX(unsigned char* input, unsigned char* output, int xSize, int ySize)
{
    unsigned char* dev_input = 0;
    unsigned char* dev_output = 0;

    unsigned int xySize = xSize * ySize;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipError_t cudaStatus_X;

    hipDeviceProp_t prop;
    int count;


    hipGetDeviceCount(&count);

    cudaStatus_X = hipSetDevice(0);
    if (cudaStatus_X != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    }


    hipEventRecord(start, 0);

    cudaStatus_X = hipMalloc((void**)&dev_input, xySize * sizeof(char));
    if (cudaStatus_X != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        //goto Error;
    }

    cudaStatus_X = hipMalloc((void**)&dev_output, xySize * sizeof(char));
    if (cudaStatus_X != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        //goto Error;
    }

    cudaStatus_X = hipMemcpy(dev_input, input, xySize, hipMemcpyHostToDevice);
    if (cudaStatus_X != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        //goto Error;
    }

    dim3 blockSize(32, 32); // 1D 블록, 1024 스레드
    dim3 gridSize(16, 16); // 1D 그리드, 1 블록

    //FlipXKernel_1thread << <blcokSize, gridSize >> > (dev_input, dev_output, xSize, ySize);
    //FlipXKernel_1024 << <blockSize, gridSize >> > (dev_input, dev_output, xSize, ySize);
    //FlipXKernel_512x512 << <blockSize, gridSize >> > (dev_input, dev_output, xSize, ySize);
    //FlipXKernel_Zblock << <blockSize, gridSize >> > (dev_input, dev_output, xSize, ySize);
    FlipXKernel_Mblock << <blockSize, gridSize >> > (dev_input, dev_output, xSize, ySize);

    cudaStatus_X = hipDeviceSynchronize();
    if (cudaStatus_X != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching invert8Kernel!\n", cudaStatus_X);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus_X = hipMemcpy(output, dev_output, xySize * sizeof(char), hipMemcpyDeviceToHost);
    if (cudaStatus_X != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }


    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float cudaElapsedTime;
    hipEventElapsedTime(&cudaElapsedTime, start, stop);
    printf("FlipX - Time for execution = %3.1f ms \n", cudaElapsedTime);
    hipEventDestroy(start);
    hipEventDestroy(stop);

Error:
    hipFree(dev_input);
    hipFree(dev_output);

    return cudaStatus_X;
}
